#include <stdio.h>
#include <stdlib.h>
#include <cula.h>
#include <cula_blas.h>
#include <cula_lapack.h>
#include <hipblas.h>

void checkStatus(culaStatus status) {
  if (!status) {
    return;
  }

  char buf[80];
  culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
  printf("%s\n", buf);
  
  culaShutdown();

  exit(EXIT_FAILURE);
}

typedef struct {
  int w;
  int h;
  culaDouble* elements;
} Matrix;

void printMatrix(Matrix M) {
  for (int i=0; i<M.h; i++) {
    for (int j=0; j<M.w; j++) {
      printf("%f ", M.elements[M.w*i+j]);
    }
    printf("\n");
  }
}

void transposeSquareMatrix(Matrix A) {
  if (A.h != A.w) {
    printf("Error in transposeSquareMatrix(): matrix is not square!\n");
    culaShutdown();
    exit(EXIT_FAILURE);
  }

  for (int i=0; i<A.h; i++) {
    for (int j=i+1; j<A.w; j++) {
      culaDouble temp = A.elements[i*A.w + j];
      A.elements[i*A.w + j] = A.elements[j*A.w + i];
      A.elements[j*A.w + i] = temp;
    }
  }

  return;
}

/**
 * Computes the eigenvalues and eigenvectors of symmetric matrix A. Returns the
 * eigenvectors in A (note that the original contents of A are destroyed) and
 * the eigenvalues in w.
 */
void eig(Matrix A, Matrix w) {
  // First check to make sure the dimensions of everything are correct.
  int N = A.h;
  if (w.h != N) {
    printf("Error in EIG: w height is not %d.\n", N);
    culaShutdown();
    exit(EXIT_FAILURE);
  }
  if (w.w != 1) {
    printf("Error in EIG: w height is not %d.\n", 1);
    culaShutdown();
    exit(EXIT_FAILURE);
  }

  // Compute the eigendecomposition
  culaStatus status = culaDsyev('V', 'U', N, A.elements, N, w.elements);
  checkStatus(status);

  // CULA stores the result in column-major order, switch it to row-major
  transposeSquareMatrix(A);

  return;
}

void eigExample() {
  int m=4;
  int k=5;

  culaStatus status;
  printf("Initializing CULA\n");
  status = culaInitialize();
  checkStatus(status);

  Matrix A, C;
  A.h = m; 
  A.w = k; 
  A.elements = (culaDouble*)malloc(A.h*A.w*sizeof(culaDouble));
  for (int i=0; i<m*k; i++) {
    A.elements[i] = (culaDouble)i;
  }
  printf("A=\n");
  printMatrix(A);

  C.h = m; 
  C.w = m; 
  C.elements = (culaDouble*)malloc(C.h*C.w*sizeof(culaDouble));

  status = culaDgemm('T', 'N', A.h, A.h, A.w, 1.0, A.elements, A.w, A.elements,
      A.w, 0.0,
      C.elements, C.h);
  checkStatus(status);

  printf("A*A' = \n");
  printMatrix(C);

  // Now compute the eigenvectors, eigenvalues
  Matrix w;
  w.h = m;
  w.w = 1;
  w.elements = (culaDouble*)malloc(w.h*w.w*sizeof(culaDouble));

  eig(C, w);

  printf("V = \n");
  printMatrix(C);
  printf("D = \n");
  printMatrix(w);

  culaShutdown();
  
  free(A.elements);
  free(C.elements);
  free(w.elements);

  printf("Done!\n");

}

int main(int argc, char* argv[]) {

  eigExample();

  return EXIT_SUCCESS;
}
