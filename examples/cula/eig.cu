#include <stdio.h>
#include <stdlib.h>
#include <cula.h>
#include <cula_blas.h>
#include <cula_lapack.h>
#include <hipblas.h>

void checkStatus(culaStatus status) {
  if (!status) {
    return;
  }

  char buf[80];
  culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
  printf("%s\n", buf);
  
  culaShutdown();

  exit(EXIT_FAILURE);
}

typedef struct {
  int w;
  int h;
  culaFloat* elements;
} Matrix;

void printMatrix(Matrix M) {
  for (int i=0; i<M.h; i++) {
    for (int j=0; j<M.w; j++) {
      printf("%f ", M.elements[M.w*i+j]);
    }
    printf("\n");
  }
}

void culaFloatExample() {
  int m=4;
  int k=4;

  culaStatus status;
  printf("Initializing CULA\n");
  status = culaInitialize();
  checkStatus(status);

  Matrix A, C;
  A.h = m; 
  A.w = k; 
  A.elements = (culaFloat*)malloc(A.h*A.w*sizeof(culaFloat));
  for (int i=0; i<m*k; i++) {
    A.elements[i] = (culaFloat)i;
  }
  printf("A=\n");
  printMatrix(A);

  C.h = m; 
  C.w = m; 
  C.elements = (culaFloat*)malloc(C.h*C.w*sizeof(culaFloat));

  status = culaSgemm('T', 'N', A.h, A.h, A.w, 1.0, A.elements, A.h, A.elements,
      A.w, 0.0,
      C.elements, C.h);
  checkStatus(status);

  printf("A*A' = \n");
  printMatrix(C);

  // Now compute the eigenvectors, eigenvalues
  Matrix w;
  w.h = m;
  w.w = 1;
  w.elements = (culaFloat*)malloc(w.h*w.w*sizeof(culaFloat));

  status = culaSsyev('V', 'U', m, C.elements, m, w.elements);
  checkStatus(status);

  printf("V = \n");
  printMatrix(C);
  printf("D = \n");
  printMatrix(w);

  culaShutdown();
  
  free(A.elements);
  free(C.elements);
  free(w.elements);

  printf("Done!\n");

}

int main(int argc, char* argv[]) {

  culaFloatExample();

  return EXIT_SUCCESS;
}
